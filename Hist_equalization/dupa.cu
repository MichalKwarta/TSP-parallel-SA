#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define CUDA_CALLABLE_MEMBER __host__ __device__
#else
#define CUDA_CALLABLE_MEMBER
#endif
#pragma hd_warning_disable
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include <iostream>
#include <stdio.h>
#include <ctime>
#include "hip/hip_runtime.h"
#include ""
using namespace std;
using namespace cv;
CUDA_CALLABLE_MEMBER class Histogram
{
public:
    Mat src;
    String filepath;
    Histogram(String path)
    {
        filepath = path;
        src = imread(path);
    }
    ~Histogram()
    {
        cout << "histogram wyczyszczony" << endl;
    }
    CUDA_CALLABLE_MEMBER void createHist()
    {
        clock_t startTime = clock();
        int ostatnislesz = filepath.find_last_of("\\");
        String filename = filepath.substr(ostatnislesz + 1, filepath.length() - 1);
        printf("Tworzenie histogramu dla: %s", (char *)&filename);
        vector<Mat> bgr_planes;
        split(src, bgr_planes);
        int histSize = 256;
        float range[] = {0, 256};
        const float *histRange = {range};
        bool uniform = true;
        bool accumulate = false;
        Mat b_hist, g_hist, r_hist;
        calcHist(&bgr_planes[0], 1, 0, Mat(), b_hist, 1, &histSize, &histRange,
                 uniform, accumulate);
        calcHist(&bgr_planes[1], 1, 0, Mat(), g_hist, 1, &histSize, &histRange,
                 uniform, accumulate);
        calcHist(&bgr_planes[2], 1, 0, Mat(), r_hist, 1, &histSize, &histRange,
                 uniform, accumulate);
        int hist_w = 512;
        int hist_h = 400;
        int bin_w = cvRound((double)hist_w / histSize);
        Mat histImage(hist_h, hist_w, CV_8UC3, Scalar(0, 0, 0));
        normalize(b_hist, b_hist, 0, histImage.rows, NORM_MINMAX, -1, Mat());
        normalize(g_hist, g_hist, 0, histImage.rows, NORM_MINMAX, -1, Mat());
        normalize(r_hist, r_hist, 0, histImage.rows, NORM_MINMAX, -1, Mat());
        for (int i = 1; i < histSize; i++)
        {
            line(histImage, Point(bin_w * (i - 1), hist_h - cvRound(b_hist.at<float>(i - 1))),
                 Point(bin_w * (i), hist_h - cvRound(b_hist.at<float>(i))),
                 Scalar(255, 0, 0), 2, 8, 0);
            line(histImage, Point(bin_w * (i - 1), hist_h - cvRound(g_hist.at<float>(i - 1))),
                 Point(bin_w * (i), hist_h - cvRound(g_hist.at<float>(i))),
                 Scalar(0, 255, 0), 2, 8, 0);
            line(histImage, Point(bin_w * (i - 1), hist_h - cvRound(r_hist.at<float>(i - 1))),
                 Point(bin_w * (i), hist_h - cvRound(r_hist.at<float>(i))),
                 Scalar(0, 0, 255), 2, 8, 0);
        }   
        namedWindow(filename, CV_WINDOW_AUTOSIZE);
        imshow(filename, histImage);
        clock_t stopTime = clock();
        double czas = (stopTime - startTime) / (double)CLOCKS_PER_SEC;
        // cout << "czas generacji histogramu: " << czas << "s" << endl;
        printf("czas generacji histogramu: %fs", &czas);
    }
};
__global__ void doit(Histogram *h)
{
    h->createHist();
}
int main(int argc, char **argv)
{
    vector<String> filenames;
    String folder = "C:\\Users\\chime\\Documents\\Visual Studio
        2015\\Projects\\histcpp\\x64\\Debug\\obrazy ";
        glob(folder, filenames);
    clock_t startTime = clock();
    for (size_t i = 0; i < filenames.size(); ++i)
    {
        Histogram *h = new Histogram(filenames[i]);
        Histogram *d_h;
        hipMalloc(&d_h, sizeof(Histogram));
        hipMemcpy(d_h, h, sizeof(Histogram), hipMemcpyHostToDevice);
        doit<<<1, 3>>>(d_h);
        hipMemcpy(h, d_h, sizeof(Histogram), hipMemcpyDeviceToHost);
        hipFree(d_h);
        delete (h);
    }
    clock_t stopTime = clock();
    double czas = (stopTime - startTime) / (double)CLOCKS_PER_SEC;
    cout << "czas generacji wszystkich histogramow: " << czas << "s" << endl;
    waitKey(0);
    getchar();
    return 0;
}