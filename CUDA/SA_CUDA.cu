#include "hip/hip_runtime.h"
#include "SA.h"
#include <vector>
#include <algorithm>
#include <time.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <omp.h>

void SA::apply()
{
	srand(time(NULL));
	std::vector<int> current = greedy();
	float currentCost = costFunction(current);

	std::vector<int> next(current);
	std::vector<int> best(current);

	int firstToSwap;
	int secondToSwap;
	double temperature = initialTemperature;
	float nextCost;
	float bestCost = currentCost;

	for (temperature = initialTemperature; temperature >= TEMP_LIMIT; temperature *= coolingRate)
	{

		for (int i = 0; i < STEPS; i++)

		{
			next = current;
			firstToSwap = rand() % size;

			do
			{

				secondToSwap = rand() % size;
			} while (firstToSwap == secondToSwap);

			// std::swap(next[firstToSwap], next[secondToSwap]);
			std::reverse(next.begin() + firstToSwap, next.begin() + secondToSwap);
			nextCost = costFunction(next);

			double difference = currentCost - nextCost;

			if (currentCost >= nextCost)
			{
				current = next;
				currentCost = nextCost;

				if (nextCost < bestCost)
				{
					bestCost = nextCost;
				}
			}
			else
			{

				if (exp((currentCost - nextCost) / temperature) > (float)rand() / RAND_MAX)
				{
					current = next;
					currentCost = nextCost;
					// break;
				}
			}
		}
	}

	std::cout << bestCost << std::endl;

	std::cout << std::endl;
}

void SA::parallelApply()
{
	srand(time(NULL));
	std::vector<int> current = greedy();
	float currentCost = costFunction(current);
	omp_set_num_threads(WORKERS);
	std::vector<int> next(current);
	std::vector<int> best(current);
	std::vector<std::vector<int>> workersPaths(WORKERS, current);
	std::vector<float> workersCosts(WORKERS,currentCost);
	std::vector<float> workersBest(WORKERS,currentCost);


	double temperature = initialTemperature;
	float nextCost;



	float bestCost = currentCost;
	int firstToSwap,secondToSwap;
	#pragma omp parallel private(current,currentCost,firstToSwap, secondToSwap, next, nextCost) firstprivate(matrix)
		{
	for (temperature = initialTemperature; temperature >= TEMP_LIMIT; temperature *= coolingRate)
	{
		
			current = workersPaths[omp_get_thread_num()];
			currentCost = workersCosts[omp_get_thread_num()];
			bestCost = workersBest[omp_get_thread_num()];
			#pragma omp for schedule(dynamic) nowait
			for (int i = 0; i < STEPS; i++)
			{
				next = current;
				firstToSwap = rand() % size;

				do
				{
					secondToSwap = rand() % size;
				} while (firstToSwap == secondToSwap);

				// std::swap(next[firstToSwap], next[secondToSwap]);
				std::reverse(next.begin() + firstToSwap, next.begin() + secondToSwap);

				nextCost = costFunction(next);

				double difference = currentCost - nextCost;

				if (currentCost >= nextCost)
				{

					current = next;
					currentCost = nextCost;
					if (nextCost <= bestCost)
					{
						bestCost = nextCost;
					}
				}
				else
				{

					if (exp((currentCost - nextCost) / temperature) > (float)rand() / RAND_MAX)
					{
						current = next;
						currentCost = nextCost;
					}
				}
			}

			workersPaths[omp_get_thread_num()] = current;
			workersCosts[omp_get_thread_num()] = currentCost;
			workersBest[omp_get_thread_num()] = bestCost;
		}


	}

	std::cout <<*std::min_element(workersBest.begin(),workersBest.end()) << std::endl;
}

float SA::costFunction(std::vector<int> path)
{
	float cost = 0;
	for (int i = 0; i < path.size() - 1; ++i)
	{
		cost += matrix[path[i]][path[i + 1]];
	}
	cost += matrix[path[size - 1]][path[0]];

	return cost;
}

std::vector<int> SA::greedy()
{
	std::vector<int> path = {0};
	std::vector<int> nodesToVisit;
	for (int i = 1; i < this->size; i++)
	{

		nodesToVisit.push_back(i);
	}
	while (nodesToVisit.size() > 0)
	{
		int min = nodesToVisit[0];
		for (int i = 1; i < nodesToVisit.size(); i++)
		{
			if (matrix[path[path.size() - 1]][nodesToVisit[i]] < matrix[path[path.size() - 1]][min])
			{
				min = nodesToVisit[i];
			}
		}
		path.push_back(min);
		nodesToVisit.erase(std::remove(nodesToVisit.begin(), nodesToVisit.end(), min), nodesToVisit.end());
	}

	return path;
}

SA::SA(float **matrixarg, int sizearg,int workers)
{
	matrix = matrixarg;
	size = sizearg;

	WORKERS = workers;
}

SA::~SA()
{
}
